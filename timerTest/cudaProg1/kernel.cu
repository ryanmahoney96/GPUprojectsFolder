#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "../highPerformanceTimer/highPerformanceTimer.h"

#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
//#include <string>

//use alt+b, u to build only this project

using namespace std;

typedef int ourVar_t;

bool allocMemory(ourVar_t** a, ourVar_t** b, ourVar_t** c, int size, int size_of_var = sizeof(ourVar_t));
void freeMemory(ourVar_t* a, ourVar_t* b, ourVar_t* c);

int main(int argc, char* argv[]) {

	cout << endl;

	srand(time(NULL));

	int size_of_array = 5;

	//press alt+shift+(arrow key) to vertical edit
	ourVar_t* a = nullptr;
	ourVar_t* b = nullptr;
	ourVar_t* c = nullptr;
	
	try {

		HighPrecisionTime htp;
		hipError_t cudaStatus;

		double htp_ret = 0.0;
		int iterations = 100;

		//if there is a command line argument, set the ouput variable to it
		if (argc > 1) {
			size_of_array = atoi(argv[1]);
		}
		else {
			cout << "No Commmand Line Argument: Size of Array Defaulting to 5" << endl;
		}

		cout << "argc: " << argc << "\nargv: " << size_of_array << endl;

		if (!allocMemory(&a, &b, &c, size_of_array)) {
			throw("Error Allocating Memory");
		}

		//"start" the timer
		htp.TimeSinceLastCall();

		//add work here

//using omp to use as many cores as possible
#pragma omp parallel for
		for (int i = 0; i < (size_of_array * iterations); i++) {


			//for (int i = 0; i < size_of_array; i++) {
			a[i / size_of_array] = rand();
			b[i / size_of_array] = rand();
			c[i / size_of_array] = 0;
			//}

			//clock the timer here / store here
			if (i % size_of_array == 0) {

				htp_ret += htp.TimeSinceLastCall();
			}
		}


		//average here
		htp_ret = htp_ret / iterations;

		/*htp.TimeSinceLastCall();

		for (int i = 0; i < size_of_array; i++) {
			c[i] = a[i] + b[i];
		}*/

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			throw("hipDeviceReset failed!");
		}

		cout << "The average run was: " << htp_ret << endl;

		/*cout << "A: " << a[0] << endl;
		cout << "B: " << b[0] << endl;
		cout << "C: " << c[0] << endl;*/

	}

	catch (char * err) {
		cerr << err << endl;
	}

	freeMemory(a, b, c);

#ifdef _WIN32 || _WIN64
	//system("pause");
#endif
	
	return 0;
}

bool allocMemory(ourVar_t** a, ourVar_t** b, ourVar_t** c, int size, int size_of_var) {

	bool retVal = true;

	int memSize = size * size_of_var;

	*a = (ourVar_t*)malloc(memSize);
	*b = (ourVar_t*)malloc(memSize);
	*c = (ourVar_t*)malloc(memSize);

	if (*a == nullptr || *b == nullptr || *c == nullptr) {
		retVal = false;
	}

	return retVal;
}

void freeMemory(ourVar_t* a, ourVar_t* b, ourVar_t* c) {
	
	if (a != nullptr) {
		free(a);
	}
	if (b != nullptr) {
		free(b);
	}
	if (c != nullptr) {
		free(c);
	}
}