#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "highPerformanceTimer.h"

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui//highgui.hpp>

#include <iostream>

using namespace cv;
using namespace std;

typedef unsigned char uchar;

const char* cpuWindow = "CPU Window";
const char* gpuWindow = "GPU Window";

Mat image;
Mat GPUImage;

__global__ void threshKernel(uchar* imageData, size_t size_of_image, int threshold)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int i = threadIdx.x;

	//change the loop / make sure not to go over the "edge"
	//for (uchar* i = imageData; i < &imageData[size_of_image]; i++) {

		if (imageData[i] > threshold) {
			imageData[i] = 255;
		}
		else {
			imageData[i] = 0;
		}

	//}

}

double CPUthreshold (int threshold, int w, int height, unsigned char* data);
double GPUthreshold(int threshold, Mat* image, Mat& renderedImage);

void on_cpu_trackbar (int cpuThresholdNum, void*);
void on_gpu_trackbar(int gpuThresholdNum, void*);


int main(int argc, char * argv[])
{
	int cpuThresholdNum = 128;
	int gpuThresholdNum = 128;

	const int threshMAX = 255;

	try {
		if (argc != 2) {
			throw("Usage: display_image ImageToLoadAndDisplay");
		}

		//read the file from the command line
		image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

		//check for an invalid input
		//Mat.data is a pointer to the image data -> if null, no image found/useable
		if (!image.data) {
			throw("Could not open or find the image");
		}

		//we want: rows, cols, channels
		cout << "The image is " << image.rows << "x" << image.cols << " in dimension" << endl;

		cvtColor(image, image, cv::COLOR_RGB2GRAY);
		cout << "Image converted to Grayscale" << endl;

		GPUImage = image.clone();

		Mat cputmp = image.clone();
		Mat gputmp = image.clone();

		double CPUTime = CPUthreshold(cpuThresholdNum, cputmp.cols, cputmp.rows, cputmp.data);
		double GPUTime = GPUthreshold(gpuThresholdNum, &cputmp, gputmp);

		//create a window for display
		namedWindow(cpuWindow, WINDOW_NORMAL);
		namedWindow(gpuWindow, WINDOW_NORMAL);

		//show the image within the display window
		imshow(cpuWindow, cputmp);
		imshow(gpuWindow, gputmp);

		//the trackbar must be placed inside a window
		//whenever the user changes the trackbar, the on_trackbar function is called
		//trackbar name, window name, an int to change(?), a maximum value(?), and a function to call
		createTrackbar("Threshold", cpuWindow, &cpuThresholdNum, threshMAX, on_cpu_trackbar);
		createTrackbar("Threshold", gpuWindow, &gpuThresholdNum, threshMAX, on_gpu_trackbar);

		on_cpu_trackbar(cpuThresholdNum, 0);
		on_gpu_trackbar(gpuThresholdNum, 0);

		//wait for the user to enter a keystroke
		cout << "The CPU took " << CPUTime << " seconds to render the threshold on the image" << endl;
		cout << "The GPU took " << GPUTime << " seconds to render the threshold on the image" << endl;

		cout << "The GPU is " << CPUTime / GPUTime << " times faster than the CPU" << endl;

		waitKey(0);
	}
	catch (char* err) {
		cout << err << endl;
	}

    return 0;
}

double CPUthreshold(int threshold, int width, int height, unsigned char* data) {
	
	//for CPU version, time this loop
	HighPrecisionTime render;
	double renderTime = 0.0;

	render.TimeSinceLastCall();

	for (uchar* i = data; i < &data[width*height]; i++) {

		if (*i > threshold) {
			*i = 255;
		}
		else {
			*i = 0;
		}

	}

	renderTime = render.TimeSinceLastCall();

	return renderTime;
}

double GPUthreshold(int threshold, Mat* img, Mat& renderedImage) {

	HighPrecisionTime render;
	HighPrecisionTime copying;
	double renderTime = 0.0;
	double copyTime = 0.0;

	//consider pass by reference
	//Mat GPUImage = (*image).clone();

	uchar* GPUImageData;
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int maxThreadsPerBlock = prop.maxThreadsPerBlock;
	int numberOfBlocks = (img->cols * img->rows) / maxThreadsPerBlock + 1;

	size_t size_of_image = img->cols * img->rows * sizeof(uchar);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		throw("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&GPUImageData, size_of_image);
	if (cudaStatus != hipSuccess) {
		throw("hipMalloc of image failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy((void*)GPUImageData, (void*)img->data, size_of_image, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		throw("hipMemcpy of image from CPU to GPU failed!");
		goto Error;
	}

	/*1024
	32401*/
	//cout << maxThreadsPerBlock << endl << numberOfBlocks << endl;

	render.TimeSinceLastCall();

	//The first argument in the execution configuration specifies the number of thread blocks in the grid, and the second specifies the number of threads in a thread block.
	threshKernel <<< numberOfBlocks, maxThreadsPerBlock >>> (GPUImageData, size_of_image, threshold);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	renderTime = render.TimeSinceLastCall();

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy((void*)renderedImage.data, GPUImageData, size_of_image, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		throw("hipMemcpy of image data from GPU to CPU failed!");
		goto Error;
	}



Error:
	hipFree(GPUImageData);
	
	return renderTime;
}

void on_cpu_trackbar(int cpuThresholdNum, void*) {

	Mat cputmp = GPUImage.clone();

	double CPUTime = CPUthreshold(cpuThresholdNum, cputmp.cols, cputmp.rows, cputmp.data);

	cout << "CPU Threshold Number: " << cpuThresholdNum << endl;

	imshow(cpuWindow, cputmp);
}

void on_gpu_trackbar(int gpuThresholdNum, void*) {

	Mat cputmp = GPUImage.clone();
	Mat gputmp = GPUImage.clone();

	double GPUTime = GPUthreshold(gpuThresholdNum, &cputmp, gputmp);

	cout << "GPU Threshold Number: " << gpuThresholdNum << endl;

	imshow(gpuWindow, gputmp);

}