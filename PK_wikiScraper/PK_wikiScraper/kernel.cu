#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""
#include "timer.h"

using namespace std;

#define USE_OMP

#if defined(_DEBUG)
#define GIGA	(1 << 20)
#else
#define GIGA	(1 << 30)
#endif

#define BMSIZE	(GIGA / 8)
#define MAX_PATTERN_LENGTH 256

__constant__ char dev_pattern[MAX_PATTERN_LENGTH];
__constant__ int dev_pattern_size;
__device__ char * dev_buffer = nullptr;
__device__ unsigned char * dev_bitmap = nullptr;

__global__ void SearchGPU_V1(char * buffer, int buffer_size, unsigned char * bitmap, int bitmap_size)
{
	//my impression: use this index to run through the buffer one char at a time
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int pIndex;

	//the pattern index
	for (pIndex = index; pIndex < dev_pattern_size; pIndex++)
	{
		char tmp = *(buffer + index + pIndex);

		if (tmp < 65 || (tmp > 90 && tmp < 97)) {
			break;
		}
		else if (tmp > 64 && tmp < 91) {
			tmp += 32;
		}

		if (tmp != *(dev_pattern + pIndex))
			break;
	}

	//if both of the words we gathered are of the same size, the pattern must match
	if (pIndex == dev_pattern_size)
	{
		int byte_number = index >> 3;
		if (byte_number < bitmap_size)
		{
			int bit_number = index % 8;

			//need atomicity
			//CUDA Atomic functions
			{
				*(bitmap + byte_number) |= (1 << bit_number);
			}
		}
	}
}

int SearchCPU_V1(char * buffer, int buffer_size, char * pattern, int pattern_size, unsigned char * bitmap, int bitmap_size)
{
	int rv = 0;

#if defined(USE_OMP)
#pragma omp parallel for
#endif
	for (int cIndex = 0; cIndex < buffer_size; cIndex++)
	{
		int pIndex;

		for (pIndex = 0; pIndex < pattern_size; pIndex++)
		{
			if (tolower(*(buffer + cIndex + pIndex)) != *(pattern + pIndex))
				break;
		}

		if (pIndex == pattern_size)
		{
			int byte_number = cIndex >> 3;
			if (byte_number < bitmap_size)
			{
				int bit_number = cIndex % 8;
#if defined(USE_OMP)
#pragma omp critical
#endif
				{
					*(bitmap + byte_number) |= (1 << bit_number);
					rv++;
				}
			}
		}
	}
	return rv;
}

/*	CStringToLower() - this function flattens a c string to all
lower case. It marches through memory until a null byte is
found. As such, some may consider this function unsafe.

By flattening the pattern, we can eliminate a tolower in
the search function - a potentially big win.

The original pointer is returned so that the function can be
used in an assignment statement.
*/
char * CStringToLower(char * s)
{
	char * rv = s;

	for (; *s != NULL; s++)
	{
		*s = tolower(*s);
	}
	return rv;
}

inline void CheckCudaAndThrow(hipError_t t, const string & message)
{
	if (t != hipSuccess)
		throw message;
}

int main(int argc, char * argv[])
{
	cout.imbue(locale(""));
	ifstream f("C:/Users/educ/Documents/enwiki-latest-abstract.xml");
	hptimer hpt;
	char * hst_buffer = nullptr;
	unsigned char * hst_bm = nullptr;
	unsigned char * chk_bm = nullptr;

#if defined(USE_OMP)
	cout << "OMP enabled on " << omp_get_max_threads() << " threads." << endl;
#endif

	try
	{
		if (argc < 2)
			throw string("First argument must be target string.");

		char * pattern = CStringToLower(argv[1]);
		int pattern_size = strlen(pattern);

		if (!f.is_open())
			throw string("File failed to open");

		hst_buffer = new char[GIGA];
		hst_bm = new unsigned char[BMSIZE]();
		chk_bm = new unsigned char[BMSIZE];

		hpt.TimeSinceLastCall();
		f.read(hst_buffer, GIGA);
		if (!f)
			throw string("Failed to read full buffer.");
		double read_time = hpt.TimeSinceLastCall();
		cout << GIGA << " bytes read from disk in " << read_time << " seconds at " << GIGA / read_time / double(1 << 30) << " GB / second." << endl;

		CheckCudaAndThrow(hipSetDevice(0), string("hipSetDevice(0) failed on line ") + to_string(__LINE__));
		CheckCudaAndThrow(hipMalloc(&dev_buffer, GIGA), string("hipMalloc failed on line ") + to_string(__LINE__));
		CheckCudaAndThrow(hipMalloc(&dev_bitmap, BMSIZE), string("hipMalloc failed on line ") + to_string(__LINE__));
		CheckCudaAndThrow(hipMemset(dev_bitmap, 0, BMSIZE), string("hipMemset failed on line ") + to_string(__LINE__));
		CheckCudaAndThrow(hipMemcpyToSymbol(HIP_SYMBOL(dev_pattern), pattern, pattern_size, 0), string("hipMemcpyToSymbol failed on line ") + to_string(__LINE__));
		CheckCudaAndThrow(hipMemcpyToSymbol(HIP_SYMBOL(dev_pattern_size), &pattern_size, sizeof(int), 0), string("hipMemcpyToSymbol failed on line ") + to_string(__LINE__));

		hpt.TimeSinceLastCall();
		CheckCudaAndThrow(hipMemcpy(dev_buffer, hst_buffer, GIGA, hipMemcpyHostToDevice), string("hipMemcpy failed on line ") + to_string(__LINE__));
		double copy_time = hpt.TimeSinceLastCall();
		cout << GIGA << " data bytes copied to GPU in " << copy_time << " seconds at " << GIGA / copy_time / double(1 << 30) << " GB / second." << endl;

		hpt.TimeSinceLastCall();
		int matches_found = SearchCPU_V1(hst_buffer, GIGA, pattern, pattern_size, hst_bm, BMSIZE);
		double time_cpu = hpt.TimeSinceLastCall();
		cout << "SearchCPU_V1 found " << matches_found << " matches in " << time_cpu << " seconds.";
		cout << " Searched " << GIGA / time_cpu / double(1 << 30) << " GB / second." << endl;

		int threads_per_block = 1024;
		dim3 grid(1024, 1024);

		hpt.TimeSinceLastCall();
		SearchGPU_V1 << <grid, threads_per_block >> >(dev_buffer, GIGA, dev_bitmap, BMSIZE);
		CheckCudaAndThrow(hipGetLastError(), string("kernel launch failed on line ") + to_string(__LINE__));
		CheckCudaAndThrow(hipDeviceSynchronize(), string("hipDeviceSynchronize() failed on line ") + to_string(__LINE__));
		double time_gpu = hpt.TimeSinceLastCall();

		CheckCudaAndThrow(hipMemcpy(chk_bm, dev_bitmap, BMSIZE, hipMemcpyDeviceToHost), string("hipMemcpy() failed on line ") + to_string(__LINE__));

		unsigned int * bm_alias = (unsigned int *)chk_bm;
		int match_count = 0;

		for (int i = 0; i < BMSIZE / sizeof(int); i++)
		{
			unsigned int c = 0;
			unsigned int v = *(bm_alias + i);
			for (c = 0; v; c++)
			{
				v &= v - 1;
			}
			match_count += c;
		}

		cout << "SearchGPU_V1 found " << match_count << " matches in " << time_gpu << " seconds.";
		cout << " Searched " << GIGA / time_gpu / double(1 << 30) << " GB / second." << endl;
		cout << endl;
		cout << "Ratio: " << time_cpu / time_gpu << " to 1" << endl;
	}
	catch (string s)
	{
		cout << s << endl;
	}

	if (dev_buffer != nullptr)
		hipFree(dev_buffer);

	if (dev_bitmap != nullptr)
		hipFree(dev_bitmap);

	if (hst_buffer != nullptr)
		delete[] hst_buffer;

	if (hst_bm != nullptr)
		delete[] hst_bm;

	if (f.is_open())
		f.close();

	hipDeviceReset();

#if defined(WIN64) || defined(WIN32)
	cout << endl;
	system("pause");
#endif

	return 0;
}